#include <iostream>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "utils.h"

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

int main(int argc, char** argv) {
    // Measure runtime
    std::chrono::high_resolution_clock::time_point start_clock; // used by all timers
    auto microsecs_allCarsTryLaneChange = std::chrono::microseconds::zero();
    auto microsecs_allCarsDriveForward = std::chrono::microseconds::zero();

    // Prepare for printing to file
    FILE* fid = argc > 1 ? fopen(argv[1], "w") : stdout; // comment out when profiling

    // Initialization
    initializeTrafficTwoLanes();
    free(numCarsInLanes); // only for init
    free(carIndicesInLanes); // only for init
    printStep(fid); // comment out when profiling

    // Simulation loop
    for (int step=0; step<NUM_STEPS; ++step) {
        printf("@ Step %d\n", step);

        // ALL CARS TRY LANE CHANGE
        start_clock = std::chrono::high_resolution_clock::now();
        //TODO
        microsecs_allCarsTryLaneChange += std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_clock);

        // ALL CARS DRIVE FORWARD
        start_clock = std::chrono::high_resolution_clock::now();
        //TODO
        microsecs_allCarsDriveForward += std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start_clock);


        printStep(fid); // comment out when profiling
    }
    printf("Num Steps: %d, Num Lanes: %d, Num Cars: %d\n", NUM_STEPS, NUM_LANES, NUM_CARS);
    printf("Num of successful lane changes = %d\n", COUNT_LANE_CHANGE);
    printf("Cumulative microseconds of allCarsTryLaneChange = %ld us\n", microsecs_allCarsTryLaneChange.count());
    printf("Cumulative microseconds of allCarsDriveForward = %ld us\n", microsecs_allCarsDriveForward.count());


    free(cars);


    return 0;
}