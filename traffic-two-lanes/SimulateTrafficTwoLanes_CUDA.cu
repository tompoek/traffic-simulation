#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>

#include "utils.h"

void checkError(hipError_t e)
{
   if (e != hipSuccess)
   {
      std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
      abort();
   }
}

void allCarsTryLaneChangeCUDA() {
    //TODO
}

__global__ 
void allCarsDriveForwardCUDA() {
    int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    //TODO
}

int main(int argc, char** argv) {
    // Measure runtime
    std::chrono::high_resolution_clock::time_point start_clock; // used by all timers
    auto microsecs_allCarsTryLaneChange = std::chrono::microseconds::zero();
    auto microsecs_allCarsDriveForward = std::chrono::microseconds::zero();

    // Prepare for printing to file
    FILE* fid = argc > 1 ? fopen(argv[1], "w") : stdout; // comment out when profiling

    // Memory allocation
    Car* carsDevice;
    checkError(hipMalloc(&carsDevice, NUM_CARS*sizeof(*carsDevice)));

    // Initialization
    //TODO
    free(numCarsInLanes);
    free(carIndicesInLanes);

    // Simulation loop
    for (int step=0; step<NUM_STEPS; ++step) {
        printf("@ Step %d\n", step);
        //TODO
    }
    printf("Num Steps: %d, Num Lanes: %d, Num Cars: %d\n", NUM_STEPS, NUM_LANES, NUM_CARS);
    printf("Num of successful lane changes = %d\n", COUNT_LANE_CHANGE);
    printf("Cumulative microseconds of allCarsTryLaneChange = %ld us\n", microsecs_allCarsTryLaneChange.count());
    printf("Cumulative microseconds of allCarsDriveForward = %ld us\n", microsecs_allCarsDriveForward.count());

    checkError(hipFree(carsDevice));
    return 0;
}
